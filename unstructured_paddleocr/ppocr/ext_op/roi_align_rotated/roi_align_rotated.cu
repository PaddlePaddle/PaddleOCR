#include "hip/hip_runtime.h"

// This code is refer from:
// https://github.com/open-mmlab/mmcv/blob/master/mmcv/ops/csrc/common/cuda/roi_align_rotated_cuda_kernel.cuh

#include <cassert>
#include <cmath>
#include <vector>

#include "paddle/extension.h"
#include <hip/hip_runtime.h>

#define CUDA_1D_KERNEL_LOOP(i, n)                                              \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);                 \
       i += blockDim.x * gridDim.x)

#define THREADS_PER_BLOCK 512

inline int GET_BLOCKS(const int N) {
  int optimal_block_num = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  int max_block_num = 4096;
  return min(optimal_block_num, max_block_num);
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600

static __inline__ __device__ double atomicAdd(double *address, double val) {
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;
  if (val == 0.0)
    return __longlong_as_double(old);
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

#endif

template <typename T>
__device__ T bilinear_interpolate(const T *input, const int height,
                                  const int width, T y, T x,
                                  const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width)
    return 0;

  if (y <= 0)
    y = 0;
  if (x <= 0)
    x = 0;

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = input[y_low * width + x_low];
  T v2 = input[y_low * width + x_high];
  T v3 = input[y_high * width + x_low];
  T v4 = input[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__device__ void
bilinear_interpolate_gradient(const int height, const int width, T y, T x,
                              T &w1, T &w2, T &w3, T &w4, int &x_low,
                              int &x_high, int &y_low, int &y_high,
                              const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0)
    y = 0;
  if (x <= 0)
    x = 0;

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = input[y_low * width + x_low];
  // T v2 = input[y_low * width + x_high];
  // T v3 = input[y_high * width + x_low];
  // T v4 = input[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

/*** Forward ***/
template <typename scalar_t>
__global__ void roi_align_rotated_cuda_forward_kernel(
    const int nthreads, const scalar_t *bottom_data,
    const scalar_t *bottom_rois, const scalar_t spatial_scale,
    const int sample_num, const bool aligned, const bool clockwise,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, scalar_t *top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const scalar_t *offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    scalar_t offset = aligned ? (scalar_t)0.5 : (scalar_t)0.0;
    scalar_t roi_center_w = offset_bottom_rois[1] * spatial_scale - offset;
    scalar_t roi_center_h = offset_bottom_rois[2] * spatial_scale - offset;
    scalar_t roi_width = offset_bottom_rois[3] * spatial_scale;
    scalar_t roi_height = offset_bottom_rois[4] * spatial_scale;
    // scalar_t theta = offset_bottom_rois[5] * M_PI / 180.0;
    scalar_t theta = offset_bottom_rois[5];
    if (clockwise) {
      theta = -theta; // If clockwise, the angle needs to be reversed.
    }
    if (!aligned) { // for backward-compatibility only
      // Force malformed ROIs to be 1x1
      roi_width = max(roi_width, (scalar_t)1.);
      roi_height = max(roi_height, (scalar_t)1.);
    }
    scalar_t bin_size_h = static_cast<scalar_t>(roi_height) /
                          static_cast<scalar_t>(pooled_height);
    scalar_t bin_size_w =
        static_cast<scalar_t>(roi_width) / static_cast<scalar_t>(pooled_width);

    const scalar_t *offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sample_num > 0)
                             ? sample_num
                             : ceilf(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sample_num > 0) ? sample_num : ceilf(roi_width / pooled_width);

    // roi_start_h and roi_start_w are computed wrt the center of RoI (x, y).
    // Appropriate translation needs to be applied after.
    scalar_t roi_start_h = -roi_height / 2.0;
    scalar_t roi_start_w = -roi_width / 2.0;
    scalar_t cosscalar_theta = cos(theta);
    scalar_t sinscalar_theta = sin(theta);

    // We do average (integral) pooling inside a bin
    const scalar_t count = max(roi_bin_grid_h * roi_bin_grid_w, 1); // e.g. = 4

    scalar_t output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) { // e.g., iy = 0, 1
      const scalar_t yy =
          roi_start_h + ph * bin_size_h +
          static_cast<scalar_t>(iy + .5f) * bin_size_h /
              static_cast<scalar_t>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const scalar_t xx = roi_start_w + pw * bin_size_w +
                            static_cast<scalar_t>(ix + .5f) * bin_size_w /
                                static_cast<scalar_t>(roi_bin_grid_w);

        // Rotate by theta (counterclockwise) around the center and translate
        scalar_t y = yy * cosscalar_theta - xx * sinscalar_theta + roi_center_h;
        scalar_t x = yy * sinscalar_theta + xx * cosscalar_theta + roi_center_w;

        scalar_t val = bilinear_interpolate<scalar_t>(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}

/*** Backward ***/
template <typename scalar_t>
__global__ void roi_align_rotated_backward_cuda_kernel(
    const int nthreads, const scalar_t *top_diff, const scalar_t *bottom_rois,
    const scalar_t spatial_scale, const int sample_num, const bool aligned,
    const bool clockwise, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, scalar_t *bottom_diff) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const scalar_t *offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not round
    scalar_t offset = aligned ? (scalar_t)0.5 : (scalar_t)0.0;
    scalar_t roi_center_w = offset_bottom_rois[1] * spatial_scale - offset;
    scalar_t roi_center_h = offset_bottom_rois[2] * spatial_scale - offset;
    scalar_t roi_width = offset_bottom_rois[3] * spatial_scale;
    scalar_t roi_height = offset_bottom_rois[4] * spatial_scale;
    // scalar_t theta = offset_bottom_rois[5] * M_PI / 180.0;
    scalar_t theta = offset_bottom_rois[5];
    if (clockwise) {
      theta = -theta; // If clockwise, the angle needs to be reversed.
    }
    if (!aligned) { // for backward-compatibility only
      // Force malformed ROIs to be 1x1
      roi_width = max(roi_width, (scalar_t)1.);
      roi_height = max(roi_height, (scalar_t)1.);
    }
    scalar_t bin_size_h = static_cast<scalar_t>(roi_height) /
                          static_cast<scalar_t>(pooled_height);
    scalar_t bin_size_w =
        static_cast<scalar_t>(roi_width) / static_cast<scalar_t>(pooled_width);

    scalar_t *offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const scalar_t *offset_top_diff = top_diff + top_offset;
    const scalar_t top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sample_num > 0)
                             ? sample_num
                             : ceilf(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sample_num > 0) ? sample_num : ceilf(roi_width / pooled_width);

    // roi_start_h and roi_start_w are computed wrt the center of RoI (x, y).
    // Appropriate translation needs to be applied after.
    scalar_t roi_start_h = -roi_height / 2.0;
    scalar_t roi_start_w = -roi_width / 2.0;
    scalar_t cosTheta = cos(theta);
    scalar_t sinTheta = sin(theta);

    // We do average (integral) pooling inside a bin
    const scalar_t count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) { // e.g., iy = 0, 1
      const scalar_t yy =
          roi_start_h + ph * bin_size_h +
          static_cast<scalar_t>(iy + .5f) * bin_size_h /
              static_cast<scalar_t>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const scalar_t xx = roi_start_w + pw * bin_size_w +
                            static_cast<scalar_t>(ix + .5f) * bin_size_w /
                                static_cast<scalar_t>(roi_bin_grid_w);

        // Rotate by theta around the center and translate
        scalar_t y = yy * cosTheta - xx * sinTheta + roi_center_h;
        scalar_t x = yy * sinTheta + xx * cosTheta + roi_center_w;

        scalar_t w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient<scalar_t>(height, width, y, x, w1, w2, w3,
                                                w4, x_low, x_high, y_low,
                                                y_high, index);

        scalar_t g1 = top_diff_this_bin * w1 / count;
        scalar_t g2 = top_diff_this_bin * w2 / count;
        scalar_t g3 = top_diff_this_bin * w3 / count;
        scalar_t g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(offset_bottom_diff + y_low * width + x_low, g1);
          atomicAdd(offset_bottom_diff + y_low * width + x_high, g2);
          atomicAdd(offset_bottom_diff + y_high * width + x_low, g3);
          atomicAdd(offset_bottom_diff + y_high * width + x_high, g4);
        } // if
      }   // ix
    }     // iy
  }       // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward

std::vector<paddle::Tensor>
RoIAlignRotatedCUDAForward(const paddle::Tensor &input,
                           const paddle::Tensor &rois, int aligned_height,
                           int aligned_width, float spatial_scale,
                           int sampling_ratio, bool aligned, bool clockwise) {

  auto num_rois = rois.shape()[0];

  auto channels = input.shape()[1];
  auto height = input.shape()[2];
  auto width = input.shape()[3];

  auto output =
      paddle::empty({num_rois, channels, aligned_height, aligned_width},
                    input.type(), paddle::GPUPlace());
  auto output_size = output.numel();

  PD_DISPATCH_FLOATING_TYPES(
      input.type(), "roi_align_rotated_cuda_forward_kernel", ([&] {
        roi_align_rotated_cuda_forward_kernel<
            data_t><<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
            output_size, input.data<data_t>(), rois.data<data_t>(),
            static_cast<data_t>(spatial_scale), sampling_ratio, aligned,
            clockwise, channels, height, width, aligned_height, aligned_width,
            output.data<data_t>());
      }));

  return {output};
}

std::vector<paddle::Tensor> RoIAlignRotatedCUDABackward(
    const paddle::Tensor &input, const paddle::Tensor &rois,
    const paddle::Tensor &grad_output, int aligned_height, int aligned_width,
    float spatial_scale, int sampling_ratio, bool aligned, bool clockwise) {

  auto num_rois = rois.shape()[0];

  auto batch_size = input.shape()[0];
  auto channels = input.shape()[1];
  auto height = input.shape()[2];
  auto width = input.shape()[3];

  auto grad_input = paddle::full({batch_size, channels, height, width}, 0.0,
                                 input.type(), paddle::GPUPlace());

  const int output_size = num_rois * aligned_height * aligned_width * channels;

  PD_DISPATCH_FLOATING_TYPES(
      grad_output.type(), "roi_align_rotated_backward_cuda_kernel", ([&] {
        roi_align_rotated_backward_cuda_kernel<
            data_t><<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
            output_size, grad_output.data<data_t>(), rois.data<data_t>(),
            spatial_scale, sampling_ratio, aligned, clockwise, channels, height,
            width, aligned_height, aligned_width, grad_input.data<data_t>());
      }));
  return {grad_input};
}