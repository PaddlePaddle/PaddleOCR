#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iterator>
#include <random>
#include <sstream>
#include <vector>

#include "fastertransformer/cuda/hipcub/hipcub.hpp"
#include "fusion_gpt_op.h"
#include "pd_traits.h"


template <paddle::DataType D>
std::vector<paddle::Tensor> gpt2_kernel(
    const paddle::Tensor& input,
    const paddle::Tensor& attn_mask,
    const paddle::Tensor& start_length,
    const paddle::Tensor& word_emb,
    const std::vector<paddle::Tensor>& self_ln_weight,
    const std::vector<paddle::Tensor>& self_ln_bias,
    const std::vector<paddle::Tensor>& self_q_weight,
    const std::vector<paddle::Tensor>& self_q_bias,
    const std::vector<paddle::Tensor>& self_k_weight,
    const std::vector<paddle::Tensor>& self_k_bias,
    const std::vector<paddle::Tensor>& self_v_weight,
    const std::vector<paddle::Tensor>& self_v_bias,
    const std::vector<paddle::Tensor>& self_out_weight,
    const std::vector<paddle::Tensor>& self_out_bias,
    const std::vector<paddle::Tensor>& ffn_ln_weight,
    const std::vector<paddle::Tensor>& ffn_ln_bias,
    const std::vector<paddle::Tensor>& ffn_inter_weight,
    const std::vector<paddle::Tensor>& ffn_inter_bias,
    const std::vector<paddle::Tensor>& ffn_out_weight,
    const std::vector<paddle::Tensor>& ffn_out_bias,
    const paddle::Tensor& decoder_ln_weight,
    const paddle::Tensor& decoder_ln_bias,
    const paddle::Tensor& positional_embedding_weight,
    const paddle::Tensor& emb_weight,
    paddle::Tensor& output_ids,
    const int& topk,
    const float& topp,
    const int& max_len,
    const int& n_head,
    const int& size_per_head,
    const int& num_layer,
    const int& bos_id,
    const int& eos_id,
    const float& temperature,
    hipblasHandle_t cublas_handle_,
    hipblasLtHandle_t cublaslt_handle_,
    hipStream_t stream) {
  auto input_dims = input.shape();
  int batch_size_ = input_dims[0];
  int start_len = input_dims[1];
  const int vocab_size = word_emb.shape()[0];

  typedef PDTraits<D> traits_;
  typedef typename traits_::DataType DataType_;
  typedef typename traits_::data_t data_t_;

  DecodingInitParam<DataType_> decoding_params;
  decoding_params.cublas_handle = cublas_handle_;
  decoding_params.cublaslt_handle = cublaslt_handle_;

  decoding_params.output_ids = output_ids.mutable_data<int>(word_emb.place());

  typedef DecoderTransformerTraits<traits_::OpType> DecodingTraits_;
  decoding_params.stream = stream;
  fastertransformer::Allocator<AllocatorType::PD> allocator_(stream);

  const int hidden_unit = size_per_head * n_head;

  TensorParallelParam tensor_parallel_param;
  LayerParallelParam layer_parallel_param;
  // TODO: multi-cards supports.
  // ncclComm_t tensor_para_nccl_comm, layer_para_nccl_comm;

  tensor_parallel_param.rank = 0;
  tensor_parallel_param.world_size = 1;
  // TODO: multi-cards supports.
  // tensor_parallel_param.nccl_comm = tensor_para_nccl_comm;
  tensor_parallel_param.local_head_num_ = n_head;
  tensor_parallel_param.local_hidden_units_ = hidden_unit;

  layer_parallel_param.rank = 0;
  layer_parallel_param.world_size = 1;
  // TODO: multi-cards supports.
  // layer_parallel_param.nccl_comm = layer_para_nccl_comm;
  layer_parallel_param.layers_per_group = num_layer;
  layer_parallel_param.local_batch_size = batch_size_;

  DecodingGpt<DecodingTraits_::OpType>* gpt_decoding;

  decoding_params.request_batch_size = batch_size_;
  decoding_params.max_input_len = start_len;
  decoding_params.request_input_len = start_len;
  decoding_params.request_output_len = max_len - start_len;

  decoding_params.d_start_ids = const_cast<int *>(input.data<int>());
  decoding_params.d_attn_mask =
      reinterpret_cast<DataType_*>(const_cast<data_t_ *>(attn_mask.data<data_t_>()));
  decoding_params.d_start_lengths = start_length.data<int>();

  gpt_decoding =
      new DecodingGpt<DecodingTraits_::OpType>(allocator_,
                                               batch_size_,
                                               max_len,
                                               n_head,
                                               size_per_head,
                                               vocab_size,
                                               num_layer,
                                               bos_id,
                                               eos_id,
                                               topk,
                                               topp,
                                               temperature,
                                               1, /*tensor_para_size*/
                                               1, /*layer_para_size*/
                                               true /*is_fuse_QKV*/);

  gpt_decoding->set_tensor_parallel_param(tensor_parallel_param);
  gpt_decoding->set_layer_parallel_param(layer_parallel_param);

  DecoderInitParam<DataType_>* params =
      new DecoderInitParam<DataType_>[num_layer];

  for (int i = 0; i < num_layer; ++i) {
    if (layer_parallel_param.is_valid(i) == false) {
      continue;
    }

    params[i].stream = stream;
    params[i].cublas_handle = cublas_handle_;
    params[i].cublaslt_handle = cublaslt_handle_;

    params[i].request_batch_size = batch_size_;
    params[i].request_max_mem_seq_len = start_len;

    params[i].self_layernorm.gamma =
        reinterpret_cast<const DataType_*>(self_ln_weight[i].data<data_t_>());
    params[i].self_layernorm.beta =
        reinterpret_cast<const DataType_*>(self_ln_bias[i].data<data_t_>());

    params[i].self_attention.query_weight.kernel =
        reinterpret_cast<const DataType_*>(self_q_weight[i].data<data_t_>());
    params[i].self_attention.query_weight.bias =
        reinterpret_cast<const DataType_*>(self_q_bias[i].data<data_t_>());
    params[i].self_attention.key_weight.kernel =
        reinterpret_cast<const DataType_*>(self_k_weight[i].data<data_t_>());
    params[i].self_attention.key_weight.bias =
        reinterpret_cast<const DataType_*>(self_k_bias[i].data<data_t_>());
    params[i].self_attention.value_weight.kernel =
        reinterpret_cast<const DataType_*>(self_v_weight[i].data<data_t_>());
    params[i].self_attention.value_weight.bias =
        reinterpret_cast<const DataType_*>(self_v_bias[i].data<data_t_>());

    params[i].self_attention.attention_output_weight.kernel =
        reinterpret_cast<const DataType_*>(self_out_weight[i].data<data_t_>());
    params[i].self_attention.attention_output_weight.bias =
        reinterpret_cast<const DataType_*>(self_out_bias[i].data<data_t_>());

    params[i].ffn_layernorm.gamma =
        reinterpret_cast<const DataType_*>(ffn_ln_weight[i].data<data_t_>());
    params[i].ffn_layernorm.beta =
        reinterpret_cast<const DataType_*>(ffn_ln_bias[i].data<data_t_>());

    params[i].ffn.intermediate_weight.kernel =
        reinterpret_cast<const DataType_*>(ffn_inter_weight[i].data<data_t_>());
    params[i].ffn.intermediate_weight.bias =
        reinterpret_cast<const DataType_*>(ffn_inter_bias[i].data<data_t_>());
    params[i].ffn.output_weight.kernel =
        reinterpret_cast<const DataType_*>(ffn_out_weight[i].data<data_t_>());
    params[i].ffn.output_weight.bias =
        reinterpret_cast<const DataType_*>(ffn_out_bias[i].data<data_t_>());
  }

  decoding_params.layernorm.gamma =
      reinterpret_cast<const DataType_*>(decoder_ln_weight.data<data_t_>());
  decoding_params.layernorm.beta =
      reinterpret_cast<const DataType_*>(decoder_ln_bias.data<data_t_>());
  decoding_params.embedding_table =
      reinterpret_cast<const DataType_*>(word_emb.data<data_t_>());
  decoding_params.embedding_kernel =
      reinterpret_cast<const DataType_*>(emb_weight.data<data_t_>());
  decoding_params.position_encoding_table = reinterpret_cast<const DataType_*>(
      positional_embedding_weight.data<data_t_>());

  gpt_decoding->forward_context(params, decoding_params);
  gpt_decoding->forward(params, decoding_params);

  delete gpt_decoding;
  delete[] params;

  return {output_ids};
}

std::vector<paddle::Tensor> GPT2CUDAForward(
    const paddle::Tensor& input,
    const paddle::Tensor& attn_mask,
    const paddle::Tensor& start_length,
    const paddle::Tensor& word_embedding,
    const std::vector<paddle::Tensor>& self_ln_weight,
    const std::vector<paddle::Tensor>& self_ln_bias,
    const std::vector<paddle::Tensor>& self_q_weight,
    const std::vector<paddle::Tensor>& self_q_bias,
    const std::vector<paddle::Tensor>& self_k_weight,
    const std::vector<paddle::Tensor>& self_k_bias,
    const std::vector<paddle::Tensor>& self_v_weight,
    const std::vector<paddle::Tensor>& self_v_bias,
    const std::vector<paddle::Tensor>& self_out_weight,
    const std::vector<paddle::Tensor>& self_out_bias,
    const std::vector<paddle::Tensor>& ffn_ln_weight,
    const std::vector<paddle::Tensor>& ffn_ln_bias,
    const std::vector<paddle::Tensor>& ffn_inter_weight,
    const std::vector<paddle::Tensor>& ffn_inter_bias,
    const std::vector<paddle::Tensor>& ffn_out_weight,
    const std::vector<paddle::Tensor>& ffn_out_bias,
    const paddle::Tensor& decoder_ln_weight,
    const paddle::Tensor& decoder_ln_bias,
    const paddle::Tensor& positional_embedding_weight,
    const paddle::Tensor& emb_weight,
    paddle::Tensor& output_ids,
    const int& topk,
    const float& topp,
    const int& max_len,
    const int& n_head,
    const int& size_per_head,
    const int& num_layer,
    const int& bos_id,
    const int& eos_id,
    const float& temperature,
    const bool& use_fp16 = false) {
  auto stream = word_embedding.stream();
  hipblasHandle_t cublas_handle_;
  hipblasCreate(&cublas_handle_);
  hipblasLtHandle_t cublaslt_handle_;
  hipblasLtCreate(&cublaslt_handle_);
  hipblasSetStream(cublas_handle_, stream);

  std::vector<paddle::Tensor> ret;

  if (use_fp16) {
    ret = gpt2_kernel<paddle::DataType::FLOAT16>(input,
                                                 attn_mask,
                                                 start_length,
                                                 word_embedding,
                                                 self_ln_weight,
                                                 self_ln_bias,
                                                 self_q_weight,
                                                 self_q_bias,
                                                 self_k_weight,
                                                 self_k_bias,
                                                 self_v_weight,
                                                 self_v_bias,
                                                 self_out_weight,
                                                 self_out_bias,
                                                 ffn_ln_weight,
                                                 ffn_ln_bias,
                                                 ffn_inter_weight,
                                                 ffn_inter_bias,
                                                 ffn_out_weight,
                                                 ffn_out_bias,
                                                 decoder_ln_weight,
                                                 decoder_ln_bias,
                                                 positional_embedding_weight,
                                                 emb_weight,
                                                 output_ids,
                                                 topk,
                                                 topp,
                                                 max_len,
                                                 n_head,
                                                 size_per_head,
                                                 num_layer,
                                                 bos_id,
                                                 eos_id,
                                                 temperature,
                                                 cublas_handle_,
                                                 cublaslt_handle_,
                                                 stream);
  } else {
    ret = gpt2_kernel<paddle::DataType::FLOAT32>(input,
                                                 attn_mask,
                                                 start_length,
                                                 word_embedding,
                                                 self_ln_weight,
                                                 self_ln_bias,
                                                 self_q_weight,
                                                 self_q_bias,
                                                 self_k_weight,
                                                 self_k_bias,
                                                 self_v_weight,
                                                 self_v_bias,
                                                 self_out_weight,
                                                 self_out_bias,
                                                 ffn_ln_weight,
                                                 ffn_ln_bias,
                                                 ffn_inter_weight,
                                                 ffn_inter_bias,
                                                 ffn_out_weight,
                                                 ffn_out_bias,
                                                 decoder_ln_weight,
                                                 decoder_ln_bias,
                                                 positional_embedding_weight,
                                                 emb_weight,
                                                 output_ids,
                                                 topk,
                                                 topp,
                                                 max_len,
                                                 n_head,
                                                 size_per_head,
                                                 num_layer,
                                                 bos_id,
                                                 eos_id,
                                                 temperature,
                                                 cublas_handle_,
                                                 cublaslt_handle_,
                                                 stream);
  }

  hipblasDestroy(cublas_handle_);
  hipblasLtDestroy(cublaslt_handle_);
  return ret;
}
