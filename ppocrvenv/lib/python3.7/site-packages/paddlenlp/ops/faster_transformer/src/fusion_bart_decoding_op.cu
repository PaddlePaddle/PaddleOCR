/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <iterator>
#include <random>
#include <sstream>
#include <vector>

#include "fastertransformer/cuda/hipcub/hipcub.hpp"
#include "fusion_bart_decoding_op.h"
#include "pd_traits.h"


template <paddle::DataType D>
std::vector<paddle::Tensor> bart_decoding_kernel(
    const paddle::Tensor& input,
    const paddle::Tensor& memory_sequence_length,
    const paddle::Tensor& word_emb,
    const std::vector<paddle::Tensor>& self_layernorm_weight,
    const std::vector<paddle::Tensor>& self_layernorm_bias,
    const std::vector<paddle::Tensor>& self_attn_query_weight,
    const std::vector<paddle::Tensor>& self_attn_query_bias,
    const std::vector<paddle::Tensor>& self_attn_key_weight,
    const std::vector<paddle::Tensor>& self_attn_key_bias,
    const std::vector<paddle::Tensor>& self_attn_value_weight,
    const std::vector<paddle::Tensor>& self_attn_value_bias,
    const std::vector<paddle::Tensor>& self_attn_output_weight,
    const std::vector<paddle::Tensor>& self_attn_output_bias,
    const std::vector<paddle::Tensor>& cross_layernorm_weight,
    const std::vector<paddle::Tensor>& cross_layernorm_bias,
    const std::vector<paddle::Tensor>& cross_attn_query_weight,
    const std::vector<paddle::Tensor>& cross_attn_query_bias,
    const std::vector<paddle::Tensor>& cross_attn_key_weight,
    const std::vector<paddle::Tensor>& cross_attn_key_bias,
    const std::vector<paddle::Tensor>& cross_attn_value_weight,
    const std::vector<paddle::Tensor>& cross_attn_value_bias,
    const std::vector<paddle::Tensor>& cross_attn_output_weight,
    const std::vector<paddle::Tensor>& cross_attn_output_bias,
    const std::vector<paddle::Tensor>& ffn_layernorm_weight,
    const std::vector<paddle::Tensor>& ffn_layernorm_bias,
    const std::vector<paddle::Tensor>& ffn_intermediate_weight,
    const std::vector<paddle::Tensor>& ffn_intermediate_bias,
    const std::vector<paddle::Tensor>& ffn_output_weight,
    const std::vector<paddle::Tensor>& ffn_output_bias,
    const paddle::Tensor& decoder_layernorm_weight,
    const paddle::Tensor& decoder_layernorm_bias,
    const paddle::Tensor& embedding_weight,
    const paddle::Tensor& embedding_bias,
    const paddle::Tensor& position_encoding_table,
    paddle::Tensor& output_ids,
    paddle::Tensor& parent_ids,
    paddle::Tensor& sequence_length,
    const std::string& decoding_strategy,
    const int& beam_size,
    const int& topk,
    const float& topp,
    const int& head_num_,
    const int& size_per_head_,
    const int& num_layer_,
    const int& start_id_,
    const int& end_id_,
    const int64_t& max_seq_len_,
    const float& beam_search_diversity_rate_,
    const float& alpha,
    const bool& early_stopping,
    hipblasHandle_t cublas_handle_,
    hipblasLtHandle_t cublaslt_handle_,
    hipStream_t stream) {
  int beam_width_ = (decoding_strategy == "beam_search" ||
                     decoding_strategy == "beam_search_v2" ||
                     decoding_strategy == "beam_search_v3")
                        ? beam_size
                        : 1;
  int candidate_num_ = (decoding_strategy == "topk_sampling" ||
                        decoding_strategy == "topp_sampling")
                           ? topk
                           : 1;
  float probability_threshold_ = (decoding_strategy == "topk_sampling" ||
                                  decoding_strategy == "topp_sampling")
                                     ? topp
                                     : 0.0;

  auto input_dims = input.shape();
  int batch_size_ = (decoding_strategy == "beam_search" ||
                     decoding_strategy == "beam_search_v2" ||
                     decoding_strategy == "beam_search_v3")
                        ? input_dims[0] / beam_width_
                        : input_dims[0];
  const int memory_max_seq_len = input_dims[1];
  const int memory_hidden_dim = input_dims[2];
  const int vocab_size = word_emb.shape()[0];

  typedef PDTraits<D> traits_;
  typedef typename traits_::DataType DataType_;
  typedef typename traits_::data_t data_t_;

  DecodingInitParam<DataType_> decoding_params;
  decoding_params.cublas_handle = cublas_handle_;
  decoding_params.cublaslt_handle = cublaslt_handle_;

  decoding_params.output_ids = output_ids.mutable_data<int>(input.place());
  decoding_params.parent_ids = parent_ids.mutable_data<int>(input.place());
  decoding_params.sequence_length =
      sequence_length.mutable_data<int>(input.place());

  typedef DecoderTransformerTraits<traits_::OpType> DecodingTraits_;
  decoding_params.stream = stream;
  fastertransformer::Allocator<AllocatorType::PD> allocator_(stream);

  decoding_params.memory_tensor =
      reinterpret_cast<const DataType_*>(input.data<data_t_>());
  decoding_params.memory_sequence_length = memory_sequence_length.data<int>();

  DecoderInitParam<DataType_>* params =
      new DecoderInitParam<DataType_>[num_layer_];

  for (int i = 0; i < num_layer_; i++) {
    params[i].stream = stream;
    params[i].cublas_handle = cublas_handle_;
    params[i].cublaslt_handle = cublaslt_handle_;

    if (decoding_strategy == "beam_search" ||
        decoding_strategy == "beam_search_v2" ||
        decoding_strategy == "beam_search_v3") {
      params[i].request_batch_size = batch_size_ * beam_width_;
      params[i].request_max_mem_seq_len = memory_max_seq_len;
    } else if (decoding_strategy == "sampling" ||
               decoding_strategy == "topk_sampling" ||
               decoding_strategy == "topp_sampling") {
      params[i].request_batch_size = batch_size_;
      params[i].request_max_mem_seq_len = memory_max_seq_len;
    }

    // self attn
    params[i].self_layernorm.gamma = reinterpret_cast<const DataType_*>(
        self_layernorm_weight[i].data<data_t_>());
    params[i].self_layernorm.beta = reinterpret_cast<const DataType_*>(
        self_layernorm_bias[i].data<data_t_>());
    // query
    params[i].self_attention.query_weight.kernel =
        reinterpret_cast<const DataType_*>(
            self_attn_query_weight[i].data<data_t_>());
    params[i].self_attention.query_weight.bias =
        reinterpret_cast<const DataType_*>(
            self_attn_query_bias[i].data<data_t_>());
    // key
    params[i].self_attention.key_weight.kernel =
        reinterpret_cast<const DataType_*>(
            self_attn_key_weight[i].data<data_t_>());
    params[i].self_attention.key_weight.bias =
        reinterpret_cast<const DataType_*>(
            self_attn_key_bias[i].data<data_t_>());
    // value
    params[i].self_attention.value_weight.kernel =
        reinterpret_cast<const DataType_*>(
            self_attn_value_weight[i].data<data_t_>());
    params[i].self_attention.value_weight.bias =
        reinterpret_cast<const DataType_*>(
            self_attn_value_bias[i].data<data_t_>());
    // out proj
    params[i].self_attention.attention_output_weight.kernel =
        reinterpret_cast<const DataType_*>(
            self_attn_output_weight[i].data<data_t_>());
    params[i].self_attention.attention_output_weight.bias =
        reinterpret_cast<const DataType_*>(
            self_attn_output_bias[i].data<data_t_>());

    // cross
    params[i].cross_layernorm.gamma = reinterpret_cast<const DataType_*>(
        cross_layernorm_weight[i].data<data_t_>());
    params[i].cross_layernorm.beta = reinterpret_cast<const DataType_*>(
        cross_layernorm_bias[i].data<data_t_>());
    // query
    params[i].cross_attention.query_weight.kernel =
        reinterpret_cast<const DataType_*>(
            cross_attn_query_weight[i].data<data_t_>());
    params[i].cross_attention.query_weight.bias =
        reinterpret_cast<const DataType_*>(
            cross_attn_query_bias[i].data<data_t_>());
    // key
    params[i].cross_attention.key_weight.kernel =
        reinterpret_cast<const DataType_*>(
            cross_attn_key_weight[i].data<data_t_>());
    params[i].cross_attention.key_weight.bias =
        reinterpret_cast<const DataType_*>(
            cross_attn_key_bias[i].data<data_t_>());
    // value
    params[i].cross_attention.value_weight.kernel =
        reinterpret_cast<const DataType_*>(
            cross_attn_value_weight[i].data<data_t_>());
    params[i].cross_attention.value_weight.bias =
        reinterpret_cast<const DataType_*>(
            cross_attn_value_bias[i].data<data_t_>());
    // out proj
    params[i].cross_attention.attention_output_weight.kernel =
        reinterpret_cast<const DataType_*>(
            cross_attn_output_weight[i].data<data_t_>());
    params[i].cross_attention.attention_output_weight.bias =
        reinterpret_cast<const DataType_*>(
            cross_attn_output_bias[i].data<data_t_>());

    // ffn
    params[i].ffn_layernorm.gamma = reinterpret_cast<const DataType_*>(
        ffn_layernorm_weight[i].data<data_t_>());
    params[i].ffn_layernorm.beta = reinterpret_cast<const DataType_*>(
        ffn_layernorm_bias[i].data<data_t_>());
    // intermediate proj
    params[i].ffn.intermediate_weight.kernel =
        reinterpret_cast<const DataType_*>(
            ffn_intermediate_weight[i].data<data_t_>());
    params[i].ffn.intermediate_weight.bias = reinterpret_cast<const DataType_*>(
        ffn_intermediate_bias[i].data<data_t_>());
    // out proj
    params[i].ffn.output_weight.kernel = reinterpret_cast<const DataType_*>(
        ffn_output_weight[i].data<data_t_>());
    params[i].ffn.output_weight.bias =
        reinterpret_cast<const DataType_*>(ffn_output_bias[i].data<data_t_>());
  }

  decoding_params.layernorm.gamma = reinterpret_cast<const DataType_*>(
      decoder_layernorm_weight.data<data_t_>());
  decoding_params.layernorm.beta = reinterpret_cast<const DataType_*>(
      decoder_layernorm_bias.data<data_t_>());
  // for embedding
  decoding_params.embedding_table =
      reinterpret_cast<const DataType_*>(word_emb.data<data_t_>());

  // for weight sharing matmul
  decoding_params.embedding_kernel =
      reinterpret_cast<const DataType_*>(embedding_weight.data<data_t_>());
  // for matmul bias
  decoding_params.embedding_bias =
      reinterpret_cast<const DataType_*>(embedding_bias.data<data_t_>());
  decoding_params.position_encoding_table = reinterpret_cast<const DataType_*>(
      position_encoding_table.data<data_t_>());

  int finished_candidate_num_ =
      ("beam_search_v3" == decoding_strategy) ? beam_width_ : beam_width_ * 2;

  if ("beam_search" == decoding_strategy) {
    DecodingBeamsearch<DecodingTraits_::OpType>* decoding_beamsearch_;
    decoding_beamsearch_ = new DecodingBeamsearch<DecodingTraits_::OpType>(
        allocator_,
        batch_size_,
        beam_width_,
        max_seq_len_,
        head_num_,
        size_per_head_,
        vocab_size,
        num_layer_,
        memory_hidden_dim,
        memory_max_seq_len,
        start_id_,
        end_id_,
        beam_search_diversity_rate_,
        true,  /*is_fuse_topk_softMax*/
        false, /*is_fuse_qkv*/
        false, /*keep_alive_beam*/
        alpha,
        false, /*normalization_before*/
        2,
        ActivationType::GELU);

    decoding_beamsearch_->forward(params, decoding_params);

    delete decoding_beamsearch_;
  } else if ("beam_search_v2" == decoding_strategy ||
             "beam_search_v3" == decoding_strategy) {
    DecodingBeamsearch<DecodingTraits_::OpType>* decoding_beamsearch_;
    decoding_beamsearch_ = new DecodingBeamsearch<DecodingTraits_::OpType>(
        allocator_,
        batch_size_,
        beam_width_,
        max_seq_len_,
        head_num_,
        size_per_head_,
        vocab_size,
        num_layer_,
        memory_hidden_dim,
        memory_max_seq_len,
        start_id_,
        end_id_,
        beam_search_diversity_rate_,
        true,  /*is_fuse_topk_softMax*/
        false, /*is_fuse_qkv*/
        true,  /*keep_alive_beam*/
        alpha,
        false, /*normalization_before*/
        2,
        ActivationType::GELU,
        false, /*pos_bias*/
        false, /*prefix_lm*/
        finished_candidate_num_,
        early_stopping);

    decoding_beamsearch_->forward(params, decoding_params);

    delete decoding_beamsearch_;
  } else if ("topk_sampling" == decoding_strategy ||
             "topp_sampling" == decoding_strategy ||
             "sampling" == decoding_strategy) {
    DecodingSampling<DecodingTraits_::OpType>* decoding_sampling_;
    decoding_sampling_ =
        new DecodingSampling<DecodingTraits_::OpType>(allocator_,
                                                      batch_size_,
                                                      max_seq_len_,
                                                      head_num_,
                                                      size_per_head_,
                                                      vocab_size,
                                                      num_layer_,
                                                      memory_hidden_dim,
                                                      memory_max_seq_len,
                                                      start_id_,
                                                      end_id_,
                                                      candidate_num_,
                                                      probability_threshold_,
                                                      false,
                                                      false,
                                                      2,
                                                      ActivationType::GELU);

    decoding_sampling_->forward(params, decoding_params);

    delete decoding_sampling_;
  } else {
    PD_THROW(
        "Only beam_search, topk_sampling and topp_sampling are supported for "
        "FasterTransformer. ");
  }
  delete[] params;

  return {output_ids, parent_ids, sequence_length};
}

std::vector<paddle::Tensor> BartDecodingCUDAForward(
    const paddle::Tensor& input,
    const paddle::Tensor& mem_seq_len,
    const paddle::Tensor& word_embedding,
    const std::vector<paddle::Tensor>& self_ln_weight,
    const std::vector<paddle::Tensor>& self_ln_bias,
    const std::vector<paddle::Tensor>& self_q_weight,
    const std::vector<paddle::Tensor>& self_q_bias,
    const std::vector<paddle::Tensor>& self_k_weight,
    const std::vector<paddle::Tensor>& self_k_bias,
    const std::vector<paddle::Tensor>& self_v_weight,
    const std::vector<paddle::Tensor>& self_v_bias,
    const std::vector<paddle::Tensor>& self_out_weight,
    const std::vector<paddle::Tensor>& self_out_bias,
    const std::vector<paddle::Tensor>& cross_ln_weight,
    const std::vector<paddle::Tensor>& cross_ln_bias,
    const std::vector<paddle::Tensor>& cross_q_weight,
    const std::vector<paddle::Tensor>& cross_q_bias,
    const std::vector<paddle::Tensor>& cross_k_weight,
    const std::vector<paddle::Tensor>& cross_k_bias,
    const std::vector<paddle::Tensor>& cross_v_weight,
    const std::vector<paddle::Tensor>& cross_v_bias,
    const std::vector<paddle::Tensor>& cross_out_weight,
    const std::vector<paddle::Tensor>& cross_out_bias,
    const std::vector<paddle::Tensor>& ffn_ln_weight,
    const std::vector<paddle::Tensor>& ffn_ln_bias,
    const std::vector<paddle::Tensor>& ffn_inter_weight,
    const std::vector<paddle::Tensor>& ffn_inter_bias,
    const std::vector<paddle::Tensor>& ffn_out_weight,
    const std::vector<paddle::Tensor>& ffn_out_bias,
    const paddle::Tensor& decoder_ln_weight,
    const paddle::Tensor& decoder_ln_bias,
    const paddle::Tensor& embedding_weight,
    const paddle::Tensor& embedding_bias,
    const paddle::Tensor& positional_embedding_weight,
    paddle::Tensor& output_ids,
    paddle::Tensor& parent_ids,
    paddle::Tensor& sequence_length,
    const std::string& decoding_strategy,
    const int& beam_size,
    const int& topk,
    const float& topp,
    const int& n_head,
    const int& size_per_head,
    const int& num_layer,
    const int& bos_id,
    const int& eos_id,
    const int64_t& max_len,
    const float& beam_search_diversity_rate,
    const float& alpha,
    const bool& early_stopping) {
  auto stream = input.stream();
  hipblasHandle_t cublas_handle_;
  hipblasCreate(&cublas_handle_);
  hipblasLtHandle_t cublaslt_handle_;
  hipblasLtCreate(&cublaslt_handle_);
  hipblasSetStream(cublas_handle_, stream);

  std::vector<paddle::Tensor> ret;

  switch (input.type()) {
    case paddle::DataType::FLOAT16: {
      ret = bart_decoding_kernel<paddle::DataType::FLOAT16>(
          input,
          mem_seq_len,
          word_embedding,
          self_ln_weight,
          self_ln_bias,
          self_q_weight,
          self_q_bias,
          self_k_weight,
          self_k_bias,
          self_v_weight,
          self_v_bias,
          self_out_weight,
          self_out_bias,
          cross_ln_weight,
          cross_ln_bias,
          cross_q_weight,
          cross_q_bias,
          cross_k_weight,
          cross_k_bias,
          cross_v_weight,
          cross_v_bias,
          cross_out_weight,
          cross_out_bias,
          ffn_ln_weight,
          ffn_ln_bias,
          ffn_inter_weight,
          ffn_inter_bias,
          ffn_out_weight,
          ffn_out_bias,
          decoder_ln_weight,
          decoder_ln_bias,
          embedding_weight,
          embedding_bias,
          positional_embedding_weight,
          output_ids,
          parent_ids,
          sequence_length,
          decoding_strategy,
          beam_size,
          topk,
          topp,
          n_head,
          size_per_head,
          num_layer,
          bos_id,
          eos_id,
          max_len,
          beam_search_diversity_rate,
          alpha,
          early_stopping,
          cublas_handle_,
          cublaslt_handle_,
          stream);
      break;
    }
    case paddle::DataType::FLOAT32: {
      ret = bart_decoding_kernel<paddle::DataType::FLOAT32>(
          input,
          mem_seq_len,
          word_embedding,
          self_ln_weight,
          self_ln_bias,
          self_q_weight,
          self_q_bias,
          self_k_weight,
          self_k_bias,
          self_v_weight,
          self_v_bias,
          self_out_weight,
          self_out_bias,
          cross_ln_weight,
          cross_ln_bias,
          cross_q_weight,
          cross_q_bias,
          cross_k_weight,
          cross_k_bias,
          cross_v_weight,
          cross_v_bias,
          cross_out_weight,
          cross_out_bias,
          ffn_ln_weight,
          ffn_ln_bias,
          ffn_inter_weight,
          ffn_inter_bias,
          ffn_out_weight,
          ffn_out_bias,
          decoder_ln_weight,
          decoder_ln_bias,
          embedding_weight,
          embedding_bias,
          positional_embedding_weight,
          output_ids,
          parent_ids,
          sequence_length,
          decoding_strategy,
          beam_size,
          topk,
          topp,
          n_head,
          size_per_head,
          num_layer,
          bos_id,
          eos_id,
          max_len,
          beam_search_diversity_rate,
          alpha,
          early_stopping,
          cublas_handle_,
          cublaslt_handle_,
          stream);
      break;
    }
    default: {
      PD_THROW(
          "NOT supported data type. "
          "Only float16 and float32 are supported. ");
      break;
    }
  }

  hipblasDestroy(cublas_handle_);
  hipblasLtDestroy(cublaslt_handle_);
  return ret;
}
