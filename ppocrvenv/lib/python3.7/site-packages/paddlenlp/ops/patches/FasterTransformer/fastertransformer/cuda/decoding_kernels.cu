#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

namespace fastertransformer {

template <typename T, bool ALIVE = false>
__global__ void init_kernel_v2(bool* finished,
                               bool* alive_finished,
                               int* sequence_length,
                               int* word_ids,
                               T* cum_log_probs,
                               const int sentence_id,
                               const int beam_width,
                               const int batch_size) {
  const bool IS_FP16 = std::is_same<T, half>::value;
  const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : 1e20f;
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < batch_size * beam_width;
       index += blockDim.x * gridDim.x) {
    finished[index] = false;
    if (index < batch_size * beam_width / 2) {
      alive_finished[index] = false;
    }
    sequence_length[index] = 0;
    if (ALIVE) {
      if (index < batch_size * beam_width / 2) word_ids[index] = sentence_id;
      cum_log_probs[index] =
          (index % beam_width == beam_width / 2) ? (T)0.0f : -MAX_T_VAL;
    } else {
      word_ids[index] = sentence_id;
      cum_log_probs[index] = (index % beam_width == 0) ? (T)0.0f : -MAX_T_VAL;
    }
  }
}

template <typename T>
void init_kernelLauncher_v2(bool* finished,
                            bool* alive_finished,
                            int* sequence_length,
                            int* word_ids,
                            T* cum_log_probs,
                            const int sentence_id,
                            const int batch_size,
                            const int beam_width,
                            hipStream_t stream) {
  dim3 grid((int)ceil(batch_size * beam_width * 1.0 / 256));
  dim3 block(256);

  init_kernel_v2<T, true><<<grid, block, 0, stream>>>(finished,
                                                      alive_finished,
                                                      sequence_length,
                                                      word_ids,
                                                      cum_log_probs,
                                                      sentence_id,
                                                      beam_width,
                                                      batch_size);
}

template <typename T>
__global__ void embedding_position_lookups_bart_kernel(
    T* from_tensor,
    const T* embedding_table,
    const T* position_encoding,
    const int* word_ids,
    const int batch_size,
    const int hidden_units) {
  // 1. lookup from embedding table
  // 2. add the position encoding
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < batch_size * hidden_units;
       index += blockDim.x * gridDim.x) {
    const int row_index = index / hidden_units;
    const int col_index = index % hidden_units;
    from_tensor[index] =
        embedding_table[word_ids[row_index] * hidden_units + col_index] +
        position_encoding[col_index];
  }
}

template <typename T>
void embedding_position_lookups_bart_kernel_launcher(T* from_tensor,
                                                     const T* embedding_table,
                                                     const T* position_encoding,
                                                     const int* word_ids,
                                                     const int batch_size,
                                                     const int hidden_units,
                                                     hipStream_t stream) {
  dim3 grid(min(batch_size, 65536));
  dim3 block(min(hidden_units, 1024));
  embedding_position_lookups_bart_kernel<T><<<grid, block, 0, stream>>>(
      from_tensor,
      embedding_table,
      position_encoding,
      word_ids,
      batch_size,
      hidden_units);
}

template <typename T>
__global__ void update_with_force_decoding_kernel(const int* trg_word,
                                                  const int* trg_length,
                                                  bool* finished,
                                                  int* word_ids,
                                                  int* sequence_length,
                                                  int* parent_ids_buf,
                                                  int* parent_ids,
                                                  int* output_ids,
                                                  T* scores,
                                                  bool keep_alive_beam,
                                                  const int batch_size,
                                                  const int beam_width,
                                                  const int max_trg_len,
                                                  const int step) {
  int bid = blockIdx.x;   // batch_size
  int tid = threadIdx.x;  // beam_width

  const T MAX_T_VAL = (sizeof(T) == 2) ? HALF_FLT_MAX : 1e20f;
  if (step <= trg_length[bid]) {
    finished[bid * beam_width + tid] = false;

    int word_id = trg_word[bid * max_trg_len + step - 1];

    if (keep_alive_beam) {
      if (tid >= beam_width / 2) {
        word_ids[bid * beam_width / 2 + tid - beam_width / 2] = word_id;
      }
    } else {
      word_ids[bid * beam_width + tid] = word_id;
    }

    output_ids[bid * beam_width + tid] = word_id;
    if (sequence_length) {
      sequence_length[bid * beam_width + tid]++;
    }

    if (parent_ids && scores) {
      if (keep_alive_beam) {
        parent_ids[bid * beam_width + tid] = bid * beam_width + beam_width / 2;
        if (tid >= beam_width / 2) {
          parent_ids_buf[bid * beam_width / 2 + tid - beam_width / 2] =
              bid * beam_width / 2;
        }

        if (tid == beam_width / 2) {
          scores[bid * beam_width + tid] = 0;
        } else {
          scores[bid * beam_width + tid] = -MAX_T_VAL;
        }
      } else {
        parent_ids[bid * beam_width + tid] = bid * beam_width;

        if (tid == 0) {
          scores[bid * beam_width + tid] = 0;
        } else {
          scores[bid * beam_width + tid] = -MAX_T_VAL;
        }
      }
    }
  }
}

template <typename T>
void update_with_force_decodingLauncher(const int* trg_word,
                                        const int* trg_length,
                                        bool* finished,
                                        int* word_ids,
                                        int* sequence_length,
                                        int* parent_ids_buf,
                                        int* parent_ids,
                                        int* output_ids,
                                        T* scores,
                                        bool keep_alive_beam,
                                        const int batch_size,
                                        const int beam_width,
                                        const int max_trg_len,
                                        const int step,
                                        hipStream_t stream) {
  if (trg_word == nullptr) {
    return;
  }

  update_with_force_decoding_kernel<<<batch_size, beam_width, 0, stream>>>(
      trg_word,
      trg_length,
      finished,
      word_ids,
      sequence_length,
      parent_ids_buf,
      parent_ids,
      output_ids,
      scores,
      keep_alive_beam,
      batch_size,
      beam_width,
      max_trg_len,
      step);
}

template <typename T>
void update_KV_cache_kernelLauncher_v2(T** key_cache,
                                       T** value_cache,
                                       const int* beam_ids,
                                       const bool* finished,
                                       const int batch_size,
                                       const int beam_width,
                                       const int head_num,
                                       const int size_per_head,
                                       const int step,
                                       const int decoder_max_seq_len,
                                       const int cache_size,
                                       const int decoder_layers,
                                       hipStream_t stream,
                                       const int memory_max_seq_len) {
  int src_id = step & 0x1;
  int tgt_id = 1 - src_id;
  int tmp_len = (memory_max_seq_len != -1) ? step + memory_max_seq_len : step;

  if (decoder_max_seq_len < 0) {
    int hidden_dim = head_num * size_per_head;
    dim3 grid(decoder_layers * batch_size * beam_width * tmp_len);
    dim3 block(min(1024, hidden_dim));
    block.x = block.x / (4 / sizeof(T));

    update_KV_cache_kernel<<<grid, block, 0, stream>>>(key_cache[src_id],
                                                       key_cache[tgt_id],
                                                       value_cache[src_id],
                                                       value_cache[tgt_id],
                                                       beam_ids,
                                                       finished,
                                                       batch_size,
                                                       beam_width,
                                                       hidden_dim,
                                                       cache_size,
                                                       tmp_len,
                                                       decoder_layers);
  } else {
    dim3 grid(batch_size * beam_width, head_num, decoder_layers);
    constexpr int block_sz = 128;
    int tmp_decoder_max_seq_len =
        (memory_max_seq_len != -1) ? (decoder_max_seq_len + memory_max_seq_len)
                                   : decoder_max_seq_len;

    update_KV_batch_major_cache_kernel<<<grid, block_sz, 0, stream>>>(
        key_cache[src_id],
        key_cache[tgt_id],
        value_cache[src_id],
        value_cache[tgt_id],
        beam_ids,
        finished,
        batch_size,
        beam_width,
        size_per_head,
        cache_size,
        tmp_len,
        tmp_decoder_max_seq_len,
        decoder_layers);
  }
}

template <typename T>
__global__ void apply_logits_mask_kernel(int vocab_size_padded,
                                         int vocab_size,
                                         int beam_width,
                                         T* log_probs,
                                         const bool* finished,
                                         const T* logits_mask = nullptr) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bbid = blockIdx.y;  // batch_size * beam_size: index

  bool finish = (finished != nullptr) ? finished[bbid] : false;

  if (!finish) {
    if (logits_mask) {
      for (int i = tid + bid * blockDim.x; i < vocab_size;
           i += blockDim.x * gridDim.x) {
        log_probs[i + bbid * vocab_size_padded] += logits_mask[i];
      }
    }
  }
}

template <typename T>
void apply_logits_mask_kernelLauncher(T* log_probs,
                                      const bool* finished,
                                      int batch_size,
                                      int beam_width,
                                      int vocab_size_padded,
                                      int vocab_size,
                                      hipStream_t stream,
                                      const T* logits_mask) {
  if (logits_mask == nullptr) return;

  dim3 block(256);
  dim3 grid((vocab_size_padded + block.x - 1) / block.x,
            beam_width * batch_size);

  apply_logits_mask_kernel<T><<<grid, block, 0, stream>>>(vocab_size_padded,
                                                          vocab_size,
                                                          beam_width,
                                                          log_probs,
                                                          finished,
                                                          logits_mask);
}

template void init_kernelLauncher_v2(bool* finished,
                                     bool* alive_finished,
                                     int* sequence_length,
                                     int* word_ids,
                                     float* cum_log_probs,
                                     const int sentence_id,
                                     const int batch_size,
                                     const int beam_width,
                                     hipStream_t stream);

template void init_kernelLauncher_v2(bool* finished,
                                     bool* alive_finished,
                                     int* sequence_length,
                                     int* word_ids,
                                     half* cum_log_probs,
                                     const int sentence_id,
                                     const int batch_size,
                                     const int beam_width,
                                     hipStream_t stream);

template void embedding_position_lookups_bart_kernel_launcher(
    float* from_tensor,
    const float* embedding_table,
    const float* position_encoding,
    const int* word_ids,
    const int batch_size,
    const int hidden_units,
    hipStream_t stream);

template void embedding_position_lookups_bart_kernel_launcher(
    half* from_tensor,
    const half* embedding_table,
    const half* position_encoding,
    const int* word_ids,
    const int batch_size,
    const int hidden_units,
    hipStream_t stream);

template void update_with_force_decodingLauncher(const int* trg_word,
                                                 const int* trg_length,
                                                 bool* finished,
                                                 int* word_ids,
                                                 int* sequence_length,
                                                 int* parent_ids_buf,
                                                 int* parent_ids,
                                                 int* output_ids,
                                                 float* scores,
                                                 bool keep_alive_beam,
                                                 const int batch_size,
                                                 const int beam_width,
                                                 const int max_trg_len,
                                                 const int step,
                                                 hipStream_t stream);

template void update_with_force_decodingLauncher(const int* trg_word,
                                                 const int* trg_length,
                                                 bool* finished,
                                                 int* word_ids,
                                                 int* sequence_length,
                                                 int* parent_ids_buf,
                                                 int* parent_ids,
                                                 int* output_ids,
                                                 half* scores,
                                                 bool keep_alive_beam,
                                                 const int batch_size,
                                                 const int beam_width,
                                                 const int max_trg_len,
                                                 const int step,
                                                 hipStream_t stream);

template void update_KV_cache_kernelLauncher_v2(float** key_cache,
                                                float** value_cache,
                                                const int* beam_ids,
                                                const bool* finished,
                                                const int batch_size,
                                                const int beam_width,
                                                const int head_num,
                                                const int size_per_head,
                                                const int step,
                                                const int decoder_max_seq_len,
                                                const int cache_size,
                                                const int decoder_layers,
                                                hipStream_t stream,
                                                const int memory_max_seq_len);

template void update_KV_cache_kernelLauncher_v2(half** key_cache,
                                                half** value_cache,
                                                const int* beam_ids,
                                                const bool* finished,
                                                const int batch_size,
                                                const int beam_width,
                                                const int head_num,
                                                const int size_per_head,
                                                const int step,
                                                const int decoder_max_seq_len,
                                                const int cache_size,
                                                const int decoder_layers,
                                                hipStream_t stream,
                                                const int memory_max_seq_len);

template void apply_logits_mask_kernelLauncher(
    float* log_probs,
    const bool* finished,
    int batch_size,
    int beam_width,
    int vocab_size_padded,
    int vocab_size,
    hipStream_t stream,
    const float* logits_mask);

template void apply_logits_mask_kernelLauncher(
    half* log_probs,
    const bool* finished,
    int batch_size,
    int beam_width,
    int vocab_size_padded,
    int vocab_size,
    hipStream_t stream,
    const half* logits_mask);

}  // end of name space fastertransformer
