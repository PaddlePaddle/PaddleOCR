#include "hip/hip_runtime.h"
// Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


// reference https://github.com/PaddlePaddle/Paddle-Inference-Demo/blob/master/python/custom-operator/custom_relu_op.cu

#include "paddle/extension.h"

template <typename data_t>
__global__ void relu_cuda_forward_kernel(const data_t* x,
                                         data_t* y,
                                         const int num) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    y[i] = max(x[i], static_cast<data_t>(0.));
  }
}

template <typename data_t>
__global__ void relu_cuda_backward_kernel(const data_t* dy,
                                          const data_t* y,
                                          data_t* dx,
                                          const int num) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = gid; i < num; i += blockDim.x * gridDim.x) {
    dx[i] = dy[i] * (y[i] > 0 ? 1. : 0.);
  }
}

std::vector<paddle::Tensor> relu_cuda_forward(const paddle::Tensor& x) {
  auto out = paddle::Tensor(paddle::PlaceType::kGPU);

  out.reshape(x.shape());
  int numel = x.size();
  int block = 512;
  int grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      x.type(), "relu_cuda_forward_kernel", ([&] {
        relu_cuda_forward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            x.data<data_t>(), out.mutable_data<data_t>(x.place()), numel);
      }));

  return {out};
}

std::vector<paddle::Tensor> relu_cuda_backward(const paddle::Tensor& x,
                                               const paddle::Tensor& out,
                                               const paddle::Tensor& grad_out) {
  auto grad_x = paddle::Tensor(paddle::PlaceType::kGPU);
  grad_x.reshape(x.shape());

  int numel = out.size();
  int block = 512;
  int grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      out.type(), "relu_cuda_backward_kernel", ([&] {
        relu_cuda_backward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            grad_out.data<data_t>(),
            out.data<data_t>(),
            grad_x.mutable_data<data_t>(x.place()),
            numel);
      }));

  return {grad_x};
}
